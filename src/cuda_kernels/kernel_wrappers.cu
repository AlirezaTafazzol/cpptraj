#include "hip/hip_runtime.h"

#include <cstdio>
#include "../DistRoutines.h"


#define BLOCKDIM 512



// device kernel def
__global__ void Action_noImage_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, int Nmols , int NAtoms, int active_size);
__global__ void Action_noImage_no_center_GPU(double *D_,double *SolventMols_,double *Solute_atoms ,double maxD, int Nmols , int NAtoms,int NSAtoms , int active_size);


//for imaging with ortho
__global__ void Action_ImageOrtho_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, double *box, int Nmols , int NAtoms, int active_size);
__global__ void Action_ImageOrtho_no_center_GPU(double *D_,double *SolventMols_,double *Solute_atoms ,double maxD, double *box, int Nmols , int NAtoms,int NSAtoms , int active_size);

//for imaging with NonOrtho
__global__ void Action_ImageNonOrtho_center_GPU(double *D_,double *maskCenter,double *SolventMols_,double maxD, double *ucell, double *recip ,int Nmols , int NAtoms, int active_size);
__global__ void Action_ImageNonOrtho_no_center_GPU(double *D_,double *SolventMols_,double *Solute_atoms ,double maxD, double *ucell, double *recip, int Nmols , int NAtoms,int NSAtoms , int active_size);

////////////////////////





void Action_Closest_Center(double *SolventMols_,double *D_, double maskCenter[3],double maxD,int  NMols, int NAtoms, float &time_gpu, ImagingType type, double* box, double* ucell, double* recip)
{


  #ifdef DEBUG_CUDA
  hipEvent_t start_event, stop_event;
  #endif

  double *devI2Ptr;
  double *devI1Ptr;
  double *devO1Ptr;
  double *boxDev;
  double *ucellDev, *recipDev;


  hipMalloc(((void **)(&devO1Ptr)),NMols * sizeof(double ));
  
  hipMalloc(((void **)(&devI1Ptr)),3 * sizeof(double ));
  hipMemcpy(devI1Ptr,maskCenter,3 * sizeof(double ),hipMemcpyHostToDevice);
  
  hipMalloc(((void **)(&devI2Ptr)),NMols * NAtoms * 3 * sizeof(double ));
  hipMemcpy(devI2Ptr,SolventMols_,NMols * NAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);



  if (type == ORTHO)
  {
    hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
    hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);
  }
  if (type == NONORTHO)
  {
    hipMalloc(((void**)(&ucellDev)), 9 * sizeof(double));
    hipMalloc(((void**)(&recipDev)), 9 * sizeof(double));
    hipMemcpy(ucellDev,ucell, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(recipDev,recip, 9 * sizeof(double), hipMemcpyHostToDevice);
  }



  int active_size  =  BLOCKDIM/NAtoms * NAtoms;
  int NBlocks = ceil(float(NMols)/ (BLOCKDIM));

  dim3 dimGrid0 = dim3(NBlocks,1);
  dim3 dimBlock0 = dim3(BLOCKDIM,1);

  #ifdef DEBUG_CUDA
  printf("NMols =  %d, NAtoms = %d\n", NMols, NAtoms); 
  printf("active_size =  %d\n", active_size);
  printf("NBlocks =  %d\n", NBlocks);
  printf("sizeof(double) = %d\n", sizeof(double));
  printf("About to launch kernel.\n");
  

  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  #endif

  if(type == NOIMAGE)
    Action_noImage_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD, NMols, NAtoms,active_size);
  else if (type == ORTHO)
    Action_ImageOrtho_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD,boxDev, NMols, NAtoms,active_size);
  else if (type == NONORTHO )
    Action_ImageNonOrtho_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr,devI1Ptr, devI2Ptr, maxD,ucellDev, recipDev, NMols, NAtoms,active_size);
  else
    printf("kernel_wrapper: error in Imagingtype\n");

  hipDeviceSynchronize();


  #ifdef DEBUG_CUDA
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&time_gpu,start_event, stop_event );

  printf("Done with kernel CUDA Kernel Time: %.2f\n", time_gpu);
  #endif

  
  hipMemcpy(D_,devO1Ptr,NMols * sizeof(double ),hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI1Ptr);
  hipFree(devI2Ptr);
  if (type == ORTHO)
    hipFree(boxDev);
  if (type == NONORTHO)
  {
    hipFree(ucellDev);
    hipFree(recipDev);
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Action_Closest_NoCenter(double *SolventMols_,double *D_, double *Solute_atoms,double maxD,int  NMols, int NAtoms,int NSAtoms, float &time_gpu, ImagingType type,double* box, double* ucell, double* recip)
{


  #ifdef DEBUG_CUDA
  hipEvent_t start_event, stop_event;
  #endif


  double *devI3Ptr;
  double *devI2Ptr;
  double *devO1Ptr;
  double *boxDev;
  double *ucellDev, *recipDev;
 
  



  hipMalloc(((void **)(&devO1Ptr)),NMols * sizeof(double ));

  hipMalloc(((void **)(&devI2Ptr)),NMols * NAtoms * 3 * sizeof(double ));
  hipMemcpy(devI2Ptr,SolventMols_,NMols * NAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);
  
  hipMalloc(((void **)(&devI3Ptr)), NSAtoms * 3 * sizeof(double ));
  hipMemcpy(devI3Ptr,Solute_atoms,NSAtoms * 3 * sizeof(double ),hipMemcpyHostToDevice);




  if (type == ORTHO)
  {
    hipMalloc(((void**)(&boxDev)), 3 * sizeof(double));
    hipMemcpy(boxDev,box, 3 * sizeof(double), hipMemcpyHostToDevice);
  }
  if (type == NONORTHO)
  {
    hipMalloc(((void**)(&ucellDev)), 9 * sizeof(double));
    hipMalloc(((void**)(&recipDev)), 9 * sizeof(double));
    hipMemcpy(ucellDev,ucell, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(recipDev,recip, 9 * sizeof(double), hipMemcpyHostToDevice);
  }

  int active_size  =  BLOCKDIM/NAtoms * NAtoms;
  int NBlocks =  ceil(NMols * NAtoms / float(active_size));

  dim3 dimGrid0 = dim3(NBlocks,1);
  dim3 dimBlock0 = dim3(BLOCKDIM,1);

   #ifdef DEBUG_CUDA
  printf("NMols =  %d, NAtoms = %d\n", NMols, NAtoms); 
  printf("active_size =  %d\n", active_size);
  printf("NBlocks =  %d\n", NBlocks);
  printf("sizeof(double) = %d\n", sizeof(double));
  printf("About to launch kernel.\n");
  

  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);
  #endif

  if(type == NOIMAGE)
    Action_noImage_no_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr, devI2Ptr,devI3Ptr, maxD, NMols, NAtoms,NSAtoms,active_size);
  else if(type == ORTHO)
    Action_ImageOrtho_no_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr, devI2Ptr,devI3Ptr, maxD, boxDev,  NMols, NAtoms,NSAtoms,active_size);
  else if (type == NONORTHO)
    Action_ImageNonOrtho_no_center_GPU<<<dimGrid0,dimBlock0>>>(devO1Ptr, devI2Ptr,devI3Ptr, maxD, ucellDev, recipDev,  NMols, NAtoms,NSAtoms,active_size);
  else
    printf("kernel_wrapper: error in type no center version\n");
  
  hipDeviceSynchronize();

  #ifdef DEBUG_CUDA
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&time_gpu,start_event, stop_event );

  printf("Done with kernel CUDA Kernel Time: %.2f\n", time_gpu);
  #endif


  
  hipMemcpy(D_,devO1Ptr,NMols * sizeof(double ),hipMemcpyDeviceToHost);
  hipFree(devO1Ptr);
  hipFree(devI2Ptr);
  hipFree(devI3Ptr);
  if (type == ORTHO)
    hipFree(boxDev);
  if (type == NONORTHO)
  {
    hipFree(ucellDev);
    hipFree(recipDev);
  }
}
